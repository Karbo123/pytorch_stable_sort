
#include <ATen/cuda/HIPContext.h>

#include <torch/extension.h>

#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#include <pybind11/pybind11.h>

namespace py = pybind11;

using f4 = float;
using f8 = double;
using i4 = int32_t;
using i8 = int64_t;


namespace stable_argsort 
{
    template <typename ValueType, typename IndexType, typename SizeType>
    void stableArgsort_cuda_kernel(ValueType *value, IndexType *index_out, SizeType length, bool increasing, hipStream_t stream)
    {
        auto policy = thrust::cuda::par.on(stream);
        if (increasing) thrust::stable_sort_by_key(policy, value, value + length, index_out, thrust::less<ValueType>());
        else thrust::stable_sort_by_key(policy, value, value + length, index_out, thrust::greater<ValueType>());   
    }

    template <typename ValueType, typename IndexType, typename SizeType>
    void stableArgsort_cpu_kernel(ValueType *value, IndexType *index_out, SizeType length, bool increasing)
    {
        auto policy = thrust::host;
        if (increasing) thrust::stable_sort_by_key(policy, value, value + length, index_out, thrust::less<ValueType>());
        else thrust::stable_sort_by_key(policy, value, value + length, index_out, thrust::greater<ValueType>());
    }

    template <typename ValueType, typename IndexType, typename SizeType>
    void stableArgsort(torch::Tensor value, torch::Tensor index_out, bool increasing)
    {
        ValueType* value_ptr     = value.data_ptr<ValueType>();
        IndexType* index_out_ptr = index_out.data_ptr<IndexType>();
        SizeType   length        = value.size(0);
        
        bool is_cuda = value.is_cuda();
        if (is_cuda)
        {
            auto device = value.get_device();
            auto stream = at::cuda::getCurrentCUDAStream();
            hipSetDevice(device);
            stableArgsort_cuda_kernel<ValueType, IndexType, SizeType>(value_ptr, index_out_ptr, length, increasing, stream);
        } else
        {
            stableArgsort_cpu_kernel<ValueType, IndexType, SizeType>(value_ptr, index_out_ptr, length, increasing);
        }
    }
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m)
{
    m.def("stable_argsort_kernel_[f4,i8,i8]", stable_argsort::stableArgsort<f4,i8,i8>, py::arg("value"), py::arg("index_out"), py::arg("increasing"));
    m.def("stable_argsort_kernel_[f8,i8,i8]", stable_argsort::stableArgsort<f8,i8,i8>, py::arg("value"), py::arg("index_out"), py::arg("increasing"));
    m.def("stable_argsort_kernel_[i4,i8,i8]", stable_argsort::stableArgsort<i4,i8,i8>, py::arg("value"), py::arg("index_out"), py::arg("increasing"));
    m.def("stable_argsort_kernel_[i8,i8,i8]", stable_argsort::stableArgsort<i8,i8,i8>, py::arg("value"), py::arg("index_out"), py::arg("increasing"));
}
